#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <iostream>
#include <chrono>
#include <time.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

extern "C++"
{
#include "../../include/CGSolver.hpp"
#include "../../include/CGSolverCuda.hpp"
}

// vector vector multiply
#define BLOCK_SIZE 128
#define SIZE 10000 // TODO: handle this better, maybe pass it as a parameter to the functions

// __global__ void vecVecMult(double *a, double *b, double *c)
// {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < SIZE)
//     {
//         c[i] = a[i] * b[i];
//     }
// }

// vector vector add
__global__ void vecVecAdd(double *a, double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < SIZE)
    {
        c[i] = a[i] + b[i];
    }
}

// vector vector subtract
__global__ void vecVecSub(double *a, double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < SIZE)
    {
        c[i] = a[i] - b[i];
    }
}

// matrix vector multiply
__global__ void matVecMult(double *A, double *x, double *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < SIZE)
    {
        double sum = 0;
        for (int j = 0; j < SIZE; j++)
        {
            sum += A[i * SIZE + j] * x[j];
        }
        y[i] = sum;
    }
}

// __global__ void matVecMult(double *A, double *b, double *out)
// {
//     __shared__ float b_shared[BLOCK_SIZE];

//     int effective_block_width;
//     if ((blockIdx.x + 1) * BLOCK_SIZE <= SIZE)
//     {
//         effective_block_width = BLOCK_SIZE;
//     }
//     else
//     {
//         // needed to avoid overflow in next row
//         effective_block_width = SIZE % BLOCK_SIZE;
//     }

//     if (threadIdx.x < effective_block_width)
//         b_shared[threadIdx.x] = b[blockIdx.x * BLOCK_SIZE + threadIdx.x];

//     __syncthreads();

//     int idy = blockIdx.y * BLOCK_SIZE + threadIdx.x;
//     float tmp_scal = 0.0;
//     // threads outside matrix dimension are not needed (vertical)
//     if (idy < SIZE)
//     {
//         for (int i = 0; i < effective_block_width; i++)
//         {
//             // take advantage of symmetric matrix for coalesced memory access
//             // tmp_scal += A[idy * SIZE + blockIdx.x * BLOCK_SIZE + i] * b_shared[i];
//             tmp_scal += b_shared[i] * A(blockIdx.x * BLOCK_SIZE + i, idy);
//         }
//         atomicAdd(out + idy, tmp_scal);
//     }
// }

__global__ void dotProduct(double *a, double *b, double *out)
{
    // each block has it's own shared_tmp of size BLOCK_SIZE
    __shared__ double shared_tmp[BLOCK_SIZE];

    // needed for atomicAdd
    if (threadIdx.x + blockDim.x * blockIdx.x == 0)
    {
        *out = 0.0;
    }

    if (blockIdx.x * blockDim.x + threadIdx.x < SIZE)
    {
        shared_tmp[threadIdx.x] = a[blockIdx.x * blockDim.x + threadIdx.x] * b[blockIdx.x * blockDim.x + threadIdx.x];
    }
    else
    {
        // needed for the reduction
        shared_tmp[threadIdx.x] = 0.0;
    }

    // reduction within block
    for (int i = blockDim.x / 2; i >= 1; i = i / 2)
    {
        // threads access memory position written by other threads so sync is needed
        __syncthreads();
        if (threadIdx.x < i)
        {
            shared_tmp[threadIdx.x] += shared_tmp[threadIdx.x + i];
        }
    }

    // atomic add the partial reduction in out
    if (threadIdx.x == 0)
    {
        atomicAdd(out, shared_tmp[0]);
    }
}

// scalar vector multiply
__global__ void scalarVecMult(double *a, double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < SIZE)
    {
        b[i] = a[i] * *c;
    }
}

// divide two scalars
__global__ void divide(double *a, double *b, double *c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0)
        *c = *a / *b;
}

__global__ void memCopy(double *in, double *out)
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < SIZE)
    {
        out[index] = in[index];
    }
}

double dot(const double *x, const double *y, size_t size)
{
    double result = 0.0;
    for (size_t i = 0; i < size; i++)
    {
        result += x[i] * y[i];
    }
    return result;
}

void axpby(double alpha, const double *x, double beta, double *y, size_t size)
{
    for (size_t i = 0; i < size; i++)
    {
        y[i] = alpha * x[i] + beta * y[i];
    }
}

// CG solver main function
void solve_cuda(double *A, double *b, double *x, size_t size, int maxIterations, double tolerance)
{
    // define dimension of the grid and block for vectors
    dim3 vecDimBlock(BLOCK_SIZE);
    dim3 vecDimGrid((SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE);
    // define dimension of the grid and block for matrices
    dim3 matDimBlock(BLOCK_SIZE);
    dim3 matDimGrid((SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
 
    // residual
    double *r = new double[size];
    // preconditioned residual
    double *p = new double[size];
    double *Ap = new double[size];
    int num_iters;
    // norm of the residual
    double r_norm = 1.0;
    double b_norm = 1.0;

    // allocate device memory
    double *d_A, *d_b, *d_x, *d_r, *d_p, *d_Ap, *d_temp;
    hipMalloc((void **)&d_A, size * size * sizeof(double));
    hipMalloc((void **)&d_b, size * sizeof(double));
    hipMalloc((void **)&d_x, size * sizeof(double));
    hipMalloc((void **)&d_r, size * sizeof(double));
    hipMalloc((void **)&d_p, size * sizeof(double));
    hipMalloc((void **)&d_Ap, size * sizeof(double));
    hipMalloc((void **)&d_temp, size * sizeof(double));

    // copy data to device
    hipMemcpy(d_A, A, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(d_r, r, size * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(d_p, p, size * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(d_Ap, Ap, size * sizeof(double), hipMemcpyHostToDevice);

    // allocate memory for five doubles, since cuda needs this type for atomicAdd, rather than double
    // (no overloading exists for variables of type double)
    double *d_alpha, *d_beta, *d_rr, *d_rr_new, *d_bb, *d_temp_scalar;
    hipMalloc((void **)&d_alpha, sizeof(double));
    hipMalloc((void **)&d_beta, sizeof(double));
    hipMalloc((void **)&d_rr, sizeof(double));
    hipMalloc((void **)&d_rr_new, sizeof(double));
    hipMalloc((void **)&d_bb, sizeof(double));
    hipMalloc((void **)&d_temp_scalar, sizeof(double));

    // initialize the residual by copying the rhs into it
    hipMemcpy(d_r, d_b, size * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(d_p, d_b, size * sizeof(double), hipMemcpyDeviceToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // calculate the dot product of the rhs, which is equal to that
    // of the residual
    dotProduct<<<vecDimGrid, vecDimBlock>>>(d_b, d_b, d_bb);
    // copy value of d_bb to d_rr
    hipMemcpy(d_rr, d_bb, sizeof(double), hipMemcpyDeviceToDevice);
    // copy value of d_bb (norm of rhs) to host
    hipMemcpy(&b_norm, d_bb, sizeof(double), hipMemcpyDeviceToHost);

    // now start the CG solver iterations
    for (num_iters = 0; num_iters < maxIterations; num_iters++)
    {
        // precA(A, p, Ap, size);
        matVecMult<<<matDimGrid, matDimBlock>>>(d_A, d_p, d_Ap);
        // Perform matrix-vector multiplication using cuBLAS gemv function
        // hipblasDgemv(handle, HIPBLAS_OP_N, size, size, &alpha, d_A, size, d_p, 1, &beta, d_Ap, 1);

        // compute new alpha coefficient
        // alpha = rr / dot(p, Ap, size);
        dotProduct<<<vecDimGrid, vecDimBlock>>>(d_p, d_Ap, d_temp_scalar);
        divide<<<1, 1>>>(d_rr, d_temp_scalar, d_alpha);

        // compute new approximate of the solution at step k+1
        // x_k+1 = x_k + alpha_k * p_k
        // axpby(alpha, p, 1.0, x, size);
        scalarVecMult<<<vecDimGrid, vecDimBlock>>>(d_p, d_temp, d_alpha);
        vecVecAdd<<<vecDimGrid, vecDimBlock>>>(d_x, d_temp, d_x);

        // compute new residual at step k+1
        // r_k+1 = r_k - alpha_k * A * p_k
        // axpby(-alpha, Ap, 1.0, r, size);
        scalarVecMult<<<vecDimGrid, vecDimBlock>>>(d_Ap, d_temp, d_alpha);
        vecVecSub<<<vecDimGrid, vecDimBlock>>>(d_r, d_temp, d_r);

        // update the 2-norm of the residual at step k+1
        // rr_new = dot(r, r, size);
        dotProduct<<<vecDimGrid, vecDimBlock>>>(d_r, d_r, d_rr_new);

        // beta_k = ||r_k+1||^2 / ||r_k||^2
        // beta = rr_new / rr;
        divide<<<1, 1>>>(d_rr_new, d_rr, d_beta);

        // update residual norm
        // rr = rr_new;
        memCopy<<<1, 1>>>(d_rr_new, d_rr);

        // stopping criterion ==> sqrt(||r||^2 / ||b||^2) < rel_error equivalent to 2-norm or euclidean norm
        // need to copy the norm of the residual back to the device
        // in order to evaluate it
        hipMemcpy(&r_norm, d_rr_new, sizeof(double), hipMemcpyDeviceToHost);

        if (std::sqrt(r_norm / b_norm) < tolerance)
        {
            break;
        }

        // compute new direction at step k+1
        // p_k+1 = r_k+1 + beta_k * p_k
        // axpby(1.0, r, beta, p, size);
        scalarVecMult<<<vecDimGrid, vecDimBlock>>>(d_p, d_temp, d_beta);
        vecVecAdd<<<vecDimGrid, vecDimBlock>>>(d_r, d_temp, d_p);
    }

    // Record stop event
    hipEventRecord(stop);
    // Synchronize to ensure that the event recording is completed
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // print the execution time
    printf("Total execution time: %f ms\n", milliseconds);

    // print the relative error and number of iterations
    printf("relative error: %e \n", std::sqrt(r_norm / b_norm));
    printf("number of iterations: %d \n", num_iters);

    // free device memory
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ap);
    hipFree(d_temp);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_rr);
    hipFree(d_rr_new);
    hipFree(d_bb);
    hipFree(d_temp_scalar);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    // free host memory
    delete[] r;
    delete[] p;
    delete[] Ap;
}

void solve_cublas(double *A, double *b, double *x, size_t size, int maxIterations, double tolerance)
{
    // define dimension of the grid and block for vectors
    dim3 vecDimBlock(BLOCK_SIZE);
    dim3 vecDimGrid((SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0;
    double beta = 0.0;

    // residual
    double *r = new double[size];
    // preconditioned residual
    double *p = new double[size];
    double *Ap = new double[size];
    int num_iters;
    // norm of the residual
    double r_norm = 1.0;
    double b_norm = 1.0;

    // allocate device memory
    double *d_A, *d_b, *d_x, *d_r, *d_p, *d_Ap, *d_temp;

    // size_t pitch;
    // hipMallocPitch((void **)&d_A, &pitch, size * sizeof(double), size);
    // hipMallocPitch((void **)&d_b, &pitch, size * sizeof(double), 1);
    // hipMallocPitch((void **)&d_x, &pitch, size * sizeof(double), 1);
    // hipMallocPitch((void **)&d_r, &pitch, size * sizeof(double), 1);
    // hipMallocPitch((void **)&d_p, &pitch, size * sizeof(double), 1);
    // hipMallocPitch((void **)&d_Ap, &pitch, size * sizeof(double), 1);
    // hipMallocPitch((void **)&d_temp, &pitch, size * sizeof(double), 1);
    
    hipMalloc((void **)&d_A, size * size * sizeof(double));
    hipMalloc((void **)&d_b, size * sizeof(double));
    hipMalloc((void **)&d_x, size * sizeof(double));
    hipMalloc((void **)&d_r, size * sizeof(double));
    hipMalloc((void **)&d_p, size * sizeof(double));
    hipMalloc((void **)&d_Ap, size * sizeof(double));
    hipMalloc((void **)&d_temp, size * sizeof(double));

    // copy data to device
    hipMemcpy(d_A, A, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(d_r, r, size * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(d_p, p, size * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(d_Ap, Ap, size * sizeof(double), hipMemcpyHostToDevice);

    // // NEEDED FOR CUBLAS DAXPY:
    double dd_alpha, dd_beta;
    // allocate device memory for dd_alpha
    hipMalloc((void **)&dd_alpha, sizeof(double));
    hipMalloc((void **)&dd_beta, sizeof(double));

    // allocate memory for five doubles, since cuda needs this type for atomicAdd, rather than double
    // (no overloading exists for variables of type double)
    double *d_alpha, *d_beta, *d_rr, *d_rr_new, *d_bb, *d_temp_scalar;
    hipMalloc((void **)&d_alpha, sizeof(double));
    hipMalloc((void **)&d_beta, sizeof(double));
    hipMalloc((void **)&d_rr, sizeof(double));
    hipMalloc((void **)&d_rr_new, sizeof(double));
    hipMalloc((void **)&d_bb, sizeof(double));
    hipMalloc((void **)&d_temp_scalar, sizeof(double));

    // initialize the residual by copying the rhs into it
    hipMemcpy(d_r, d_b, size * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(d_p, d_b, size * sizeof(double), hipMemcpyDeviceToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // calculate the dot product of the rhs, which is equal to that
    // of the residual
    dotProduct<<<vecDimGrid, vecDimBlock>>>(d_b, d_b, d_bb);
    // copy value of d_bb to d_rr
    hipMemcpy(d_rr, d_bb, sizeof(double), hipMemcpyDeviceToDevice);
    // copy value of d_bb (norm of rhs) to host
    hipMemcpy(&b_norm, d_bb, sizeof(double), hipMemcpyDeviceToHost);

    // now start the CG solver iterations
    for (num_iters = 0; num_iters < maxIterations; num_iters++)
    {
        // Perform matrix-vector multiplication using cuBLAS gemv function  
        hipblasDgemv(handle, HIPBLAS_OP_N, size, size, &alpha, d_A, size, d_p, 1, &beta, d_Ap, 1);

        // compute new alpha coefficient
        // alpha = rr / dot(p, Ap, size);
        dotProduct<<<vecDimGrid, vecDimBlock>>>(d_p, d_Ap, d_temp_scalar);
        // hipblasDdot(handle, SIZE, d_p, 1, d_Ap, 1, d_temp_scalar);
        divide<<<1, 1>>>(d_rr, d_temp_scalar, d_alpha);

        // compute new approximate of the solution at step k+1
        // x_k+1 = x_k + alpha_k * p_k
        // axpby(alpha, p, 1.0, x, size);
        scalarVecMult<<<vecDimGrid, vecDimBlock>>>(d_p, d_temp, d_alpha);
        vecVecAdd<<<vecDimGrid, vecDimBlock>>>(d_x, d_temp, d_x);
        // copy value of d_alpha to dd_alpha
        // hipMemcpy(&dd_alpha, d_alpha, sizeof(double), hipMemcpyDeviceToDevice);
        // hipblasDaxpy(handle, SIZE, &dd_alpha, d_p, 1, d_x, 1);

        // compute new residual at step k+1
        // r_k+1 = r_k - alpha_k * A * p_k
        // axpby(-alpha, Ap, 1.0, r, size);
        scalarVecMult<<<vecDimGrid, vecDimBlock>>>(d_Ap, d_temp, d_alpha);
        vecVecSub<<<vecDimGrid, vecDimBlock>>>(d_r, d_temp, d_r);
        // hipblasDaxpy(handle, SIZE, &dd_alpha, d_Ap, 1, d_r, 1);

        // update the 2-norm of the residual at step k+1
        // rr_new = dot(r, r, size);
        dotProduct<<<vecDimGrid, vecDimBlock>>>(d_r, d_r, d_rr_new);

        // beta_k = ||r_k+1||^2 / ||r_k||^2
        // beta = rr_new / rr;
        divide<<<1, 1>>>(d_rr_new, d_rr, d_beta);

        // update residual norm
        // rr = rr_new;
        memCopy<<<1, 1>>>(d_rr_new, d_rr);

        // stopping criterion ==> sqrt(||r||^2 / ||b||^2) < rel_error equivalent to 2-norm or euclidean norm
        // need to copy the norm of the residual back to the device
        // in order to evaluate it
        hipMemcpy(&r_norm, d_rr_new, sizeof(double), hipMemcpyDeviceToHost);

        if (std::sqrt(r_norm / b_norm) < tolerance)
        {
            break;
        }

        // compute new direction at step k+1
        // p_k+1 = r_k+1 + beta_k * p_k
        // axpby(1.0, r, beta, p, size);
        scalarVecMult<<<vecDimGrid, vecDimBlock>>>(d_p, d_temp, d_beta);
        vecVecAdd<<<vecDimGrid, vecDimBlock>>>(d_r, d_temp, d_p);
        // copy value of d_beta to dd_beta
        // hipMemcpy(&dd_beta, d_beta, sizeof(double), hipMemcpyDeviceToDevice);
        // hipblasDaxpy(handle, SIZE, &dd_beta, d_p, 1, d_r, 1);
    }

    // Record stop event
    hipEventRecord(stop);
    // Synchronize to ensure that the event recording is completed
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // print the execution time
    printf("Total execution time: %f ms\n", milliseconds);

    // print the relative error and number of iterations
    printf("relative error: %e \n", std::sqrt(r_norm / b_norm));
    printf("number of iterations: %d \n", num_iters);

    // free device memory
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ap);
    hipFree(d_temp);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_rr);
    hipFree(d_rr_new);
    hipFree(d_bb);
    hipFree(d_temp_scalar);
    hipFree(&dd_alpha);
    hipFree(&dd_beta);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    // free host memory
    delete[] r;
    delete[] p;
    delete[] Ap;
}

// main function
void kernel_wrapper(double *matrix, double *rhs, double *sol, size_t size, int max_iters, double rel_error)
{
    printf("Calling kernel!\n");
    // solve_cuda(matrix, rhs, sol, size, max_iters, rel_error);
    solve_cublas(matrix, rhs, sol, size, max_iters, rel_error);
}